/* internal includes */
#include <cuda_trie.cuh>

// ------------------------------
// implementations
// ------------------------------

cuda_Trie *cuda_Trie::DumpToGpu() const {
    cuda_Trie *d_trie;
    CUDA_ASSERT_SUCCESS(hipMallocAsync(&d_trie, sizeof(cuda_Trie), g_cudaGlobalConf->asyncStream));
    CUDA_ASSERT_SUCCESS(
        hipMemcpyAsync(d_trie, this, sizeof(cuda_Trie), hipMemcpyHostToDevice, g_cudaGlobalConf->asyncStream));

    return d_trie;
}
