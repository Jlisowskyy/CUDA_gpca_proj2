/* internal includes */
#include <defines.cuh>

/* external includes */
#include <format>
#include <iostream>

void AssertSuccess(const hipError_t error, const char *file, const int line) {
    TraceError(error, file, line);

    if (error != hipSuccess) {
        exit(EXIT_FAILURE);
    }
}

bool TraceError(const hipError_t error, const char *file, const int line) {
    if (error != hipSuccess) {
        std::cerr << std::format("CUDA Error at {}:{} - {}\n", file, line, hipGetErrorString(error)) << std::endl;
    }

    return error != hipSuccess;
}
