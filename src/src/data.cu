#include "hip/hip_runtime.h"
#include <data.cuh>

std::tuple<cuda_Solution *, uint32_t *> cuda_Solution::DumpToGPU(const size_t num_solutions) {
    uint32_t *d_data{};
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_data, GetMemBlockSize(num_solutions)));
    CUDA_ASSERT_SUCCESS(hipMemset(d_data, INT_MAX, GetMemBlockSize(num_solutions)));

    cuda_Solution *d_solutions{};
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_solutions, sizeof(cuda_Solution)));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_solutions->_data, &d_data, sizeof(uint32_t *), hipMemcpyHostToDevice));

    return {d_solutions, d_data};
}

cuda_Allocator::cuda_Allocator(const uint32_t max_nodes, const uint32_t max_threads,
                               const uint32_t max_node_per_thread): _max_nodes(max_nodes),
                                                                    _max_threads(max_threads),
                                                                    _max_node_per_thread(max_node_per_thread) {
    _data = new Node_[max_nodes + 1];
    _node_counters = new uint32_t[max_threads];
    _thread_nodes = new uint32_t[max_threads];

    uint32_t last_node = 1;
    /* prepare data for nodes */
    for (uint32_t t_idx = 0; t_idx < max_threads; ++t_idx) {
        uint32_t t_node_idx = _thread_nodes[t_idx] = last_node++;
        _node_counters[t_idx] = max_node_per_thread;

        /* prepare nodes */
        assert(last_node <= max_nodes && "DETECTED OVERFLOW");
        for (uint32_t node_idx = 0; node_idx < max_node_per_thread; ++node_idx) {
            _data[t_node_idx].seq_idx = UINT32_MAX;
            _data[t_node_idx].next[0] = last_node;
            t_node_idx = last_node++;

            assert(last_node <= max_nodes && "DETECTED OVERFLOW");
        }
    }

    _last_node = last_node;
}

cuda_Allocator *cuda_Allocator::DumpToGPU() const {
    cuda_Allocator *d_allocator;

    /* copy object */
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_allocator, sizeof(cuda_Allocator)));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_allocator, this, sizeof(cuda_Allocator), hipMemcpyHostToDevice));

    /* copy data */
    Node_ *d_data;

    CUDA_ASSERT_SUCCESS(hipMalloc(&d_data, (_max_nodes + 1) * sizeof(Node_)));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_data, _data, (_max_nodes + 1) * sizeof(Node_), hipMemcpyHostToDevice));

    /* copy node_counters */
    uint32_t *d_node_counters;
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_node_counters, _max_threads * sizeof(uint32_t)));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_node_counters, _node_counters, _max_threads * sizeof(uint32_t),
        hipMemcpyHostToDevice));

    /* copy thread_nodes */
    uint32_t *d_thread_nodes;
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_thread_nodes, _max_threads * sizeof(uint32_t)));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_thread_nodes, _thread_nodes, _max_threads * sizeof(uint32_t),
        hipMemcpyHostToDevice));

    uint32_t *d_idxes;
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_idxes, _max_threads * sizeof(uint32_t)));

    /* update object */
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_allocator->_data, &d_data, sizeof(Node_ *), hipMemcpyHostToDevice));
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_allocator->_node_counters, &d_node_counters, sizeof(uint32_t *),
        hipMemcpyHostToDevice));
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_allocator->_thread_nodes, &d_thread_nodes, sizeof(uint32_t *),
        hipMemcpyHostToDevice));
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_allocator->_d_idxes, &d_idxes, sizeof(uint32_t *), hipMemcpyHostToDevice));

    return d_allocator;
}

void cuda_Allocator::DeallocGPU(cuda_Allocator *d_allocator) {
    Node_ *d_data;
    uint32_t *d_node_counters;
    uint32_t *d_thread_nodes;

    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_data, &d_allocator->_data, sizeof(Node_ *), hipMemcpyDeviceToHost));
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_node_counters, &d_allocator->_node_counters, sizeof(uint32_t *),
        hipMemcpyDeviceToHost));
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_thread_nodes, &d_allocator->_thread_nodes, sizeof(uint32_t *),
        hipMemcpyDeviceToHost));

    CUDA_ASSERT_SUCCESS(hipFree(d_data));
    CUDA_ASSERT_SUCCESS(hipFree(d_node_counters));
    CUDA_ASSERT_SUCCESS(hipFree(d_thread_nodes));
    CUDA_ASSERT_SUCCESS(hipFree(d_allocator));
}

// ------------------------------
// GPU functions
// ------------------------------

void cuda_Allocator::Consolidate(const uint32_t t_idx) {
    /* wait for all threads to finish using allocator */
    __syncthreads();

    /* first thread will update global data */
    if (t_idx == 0) {
        _prepareIdxes();
    }

    /* wait for first thread to update global data */
    __syncthreads();

    /* each of threads will clean up its allocator space */
    _cleanUpOwnSpace(t_idx);

    /* No need to wait as each thread is working on its own space */
}

void cuda_Allocator::_prepareIdxes() {
    uint32_t last_node = _last_node;

    for (uint32_t t_idx = 0; t_idx < _max_threads; ++t_idx) {
        _d_idxes[t_idx] = last_node;

        last_node += (_max_node_per_thread - _node_counters[t_idx]);
    }

    _last_node = last_node;
}

void cuda_Allocator::_cleanUpOwnSpace(const uint32_t t_idx) {
}

// ------------------------------
// Cuda data functions
// ------------------------------

cuda_Data::cuda_Data(const BinSequencePack &pack): cuda_Data(pack.sequences.size(),
                                                             (pack.max_seq_size_bits + 31) / 32) {
    static constexpr uint64_t kBitMask32 = ~static_cast<uint32_t>(0);

    for (size_t seq_idx = 0; seq_idx < pack.sequences.size(); ++seq_idx) {
        const auto &sequence = pack.sequences[seq_idx];
        auto fetcher = (*this)[seq_idx];

        /* user dwords for better performance */
        for (size_t qword_idx = 0; qword_idx < sequence.GetSizeWords(); ++qword_idx) {
            const uint64_t qword = sequence.GetWord(qword_idx);
            const size_t dword_idx = qword_idx * 2;
            const uint32_t lo = qword & kBitMask32;
            const uint32_t hi = (qword >> 32) & kBitMask32;

            fetcher.GetWord(dword_idx) = lo;
            fetcher.GetWord(dword_idx + 1) = hi;
        }
    }
}

cuda_Data * cuda_Data::DumpToGPU() const {
    /* allocate manager object */
    cuda_Data *d_data;

    CUDA_ASSERT_SUCCESS(hipMalloc(&d_data, sizeof(cuda_Data)));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_data, this, sizeof(cuda_Data), hipMemcpyHostToDevice));

    /* allocate data itself */
    uint32_t *d_data_data;

    const size_t data_size = _num_sequences_padded32 * (_max_sequence_length + 1) * sizeof(uint32_t);
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_data_data, data_size));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_data_data, _data, data_size, hipMemcpyHostToDevice));

    /* update manager object */
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_data->_data, &d_data_data, sizeof(uint32_t *), hipMemcpyHostToDevice));

    return d_data;
}

uint32_t * cuda_Data::GetDataPtrHost(const cuda_Data *d_data) {
    uint32_t *ptr;
    CUDA_ASSERT_SUCCESS(hipMemcpy(&ptr, d_data->_data, sizeof(uint32_t *), hipMemcpyDeviceToHost));
    return ptr;
}

void cuda_Data::DeallocGPU(cuda_Data *d_data) {
    uint32_t *d_data_ptr = GetDataPtrHost(d_data);
    CUDA_ASSERT_SUCCESS(hipFree(d_data_ptr));
    CUDA_ASSERT_SUCCESS(hipFree(d_data));
}
