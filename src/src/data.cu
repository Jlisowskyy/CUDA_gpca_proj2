/* internal includes */
#include <data.cuh>

/* external includes */
#include <barrier>
#include <iostream>
#include <memory>

std::tuple<cuda_Solution *, uint32_t *> cuda_Solution::DumpToGPU(const size_t num_solutions) {
    uint32_t *d_data{};
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_data, GetMemBlockSize(num_solutions)));
    CUDA_ASSERT_SUCCESS(hipMemset(d_data, UINT32_MAX, GetMemBlockSize(num_solutions)));

    cuda_Solution *d_solutions{};
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_solutions, sizeof(cuda_Solution)));
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_solutions->_data, &d_data, sizeof(uint32_t *), hipMemcpyHostToDevice));

    return {d_solutions, d_data};
}

cuda_Allocator::cuda_Allocator(const uint32_t max_nodes, const uint32_t max_threads,
                               const uint32_t max_node_per_thread): _max_nodes(max_nodes),
                                                                    _max_threads(max_threads),
                                                                    _max_node_per_thread(max_node_per_thread) {
    _node_counters = new uint32_t[max_threads];
    _thread_nodes = new uint32_t[max_threads];
    _thread_tails = new uint32_t[max_threads];
    _idxes = new uint32_t[max_threads];

    const size_t helper_size = (4 * max_threads * sizeof(uint32_t));
    const size_t expected_total_mem_used = helper_size + ((max_nodes + 1) * sizeof(Node_));
    size_t free_mem;

    CUDA_ASSERT_SUCCESS(hipMemGetInfo(&free_mem, nullptr));

    size_t total_mem_used = expected_total_mem_used;
    if (expected_total_mem_used > free_mem) {
        total_mem_used = 8 * free_mem / 10;
        const size_t mem_for_nodes = total_mem_used - helper_size;
        _max_nodes = mem_for_nodes / sizeof(Node_) - 1;

        std::cout << "Reduced number of nodes to " << _max_nodes << " from " << max_nodes << std::endl;
    }

    _data = new Node_[_max_nodes + 1]{};

    std::cout << "Allocated " << total_mem_used / (1024 * 1024) << " mega bytes for cuda_Allocator" <<
            std::endl;
    std::cout << "Expected was " << expected_total_mem_used / (1024 * 1024) << " mega bytes" << std::endl;

    uint32_t last_node = 1;
    /* prepare data for nodes */
    for (uint32_t t_idx = 0; t_idx < max_threads; ++t_idx) {
        uint32_t t_node_idx = _thread_nodes[t_idx] = last_node++;
        _node_counters[t_idx] = max_node_per_thread + 1;

        /* prepare nodes */
        assert(last_node <= max_nodes && "DETECTED OVERFLOW");
        /* Contains one extra node for the thread serving as a sentinel to never find an empty list */
        for (uint32_t node_idx = 0; node_idx < max_node_per_thread; ++node_idx) {
            assert(_data[t_node_idx].next[0] == 0 && _data[t_node_idx].next[1] == 0 && _data[t_node_idx].seq_idx == 0);

            _data[t_node_idx].seq_idx = UINT32_MAX;
            _data[t_node_idx].next[0] = last_node;
            _data[t_node_idx].next[1] = 0;
            t_node_idx = last_node++;

            assert(last_node <= max_nodes && "DETECTED OVERFLOW");
        }

        /* ensure that last node is cleaned */
        assert(_data[t_node_idx].next[0] == 0 && _data[t_node_idx].next[1] == 0 && _data[t_node_idx].seq_idx == 0);
        _data[t_node_idx].next[0] = _data[t_node_idx].next[1] = 0;
        _data[t_node_idx].seq_idx = UINT32_MAX;

        _thread_tails[t_idx] = t_node_idx;
    }

    _last_node = last_node;
}

cuda_Allocator *cuda_Allocator::DumpToGPU() const {
    cuda_Allocator *d_allocator;

    /* copy object */
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_allocator, sizeof(cuda_Allocator)));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_allocator, this, sizeof(cuda_Allocator), hipMemcpyHostToDevice));

    /* copy data */
    Node_ *d_data;

    CUDA_ASSERT_SUCCESS(hipMalloc(&d_data, (_max_nodes + 1) * sizeof(Node_)));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_data, _data, (_max_nodes + 1) * sizeof(Node_), hipMemcpyHostToDevice));

    /* copy node_counters */
    uint32_t *d_node_counters;
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_node_counters, _max_threads * sizeof(uint32_t)));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_node_counters, _node_counters, _max_threads * sizeof(uint32_t),
        hipMemcpyHostToDevice));

    /* copy thread_nodes */
    uint32_t *d_thread_nodes;
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_thread_nodes, _max_threads * sizeof(uint32_t)));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_thread_nodes, _thread_nodes, _max_threads * sizeof(uint32_t),
        hipMemcpyHostToDevice));

    /* copy idxes */
    uint32_t *d_idxes;
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_idxes, _max_threads * sizeof(uint32_t)));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_idxes, _idxes, _max_threads * sizeof(uint32_t), hipMemcpyHostToDevice));

    /* copy thread_tails */
    uint32_t *d_thread_tails;
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_thread_tails, _max_threads * sizeof(uint32_t)));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_thread_tails, _thread_tails, _max_threads * sizeof(uint32_t),
        hipMemcpyHostToDevice));

    /* update object */
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_allocator->_data, &d_data, sizeof(Node_ *), hipMemcpyHostToDevice));
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_allocator->_node_counters, &d_node_counters, sizeof(uint32_t *),
        hipMemcpyHostToDevice));
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_allocator->_thread_nodes, &d_thread_nodes, sizeof(uint32_t *),
        hipMemcpyHostToDevice));
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_allocator->_idxes, &d_idxes, sizeof(uint32_t *), hipMemcpyHostToDevice));
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_allocator->_thread_tails, &d_thread_tails, sizeof(uint32_t *),
        hipMemcpyHostToDevice));

    return d_allocator;
}

void cuda_Allocator::DeallocGPU(cuda_Allocator *d_allocator) {
    Node_ *d_data;
    uint32_t *d_node_counters;
    uint32_t *d_thread_nodes;
    uint32_t *d_idxes;
    uint32_t *d_thread_tails;

    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_data, &d_allocator->_data, sizeof(Node_ *), hipMemcpyDeviceToHost));
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_node_counters, &d_allocator->_node_counters, sizeof(uint32_t *),
        hipMemcpyDeviceToHost));
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_thread_nodes, &d_allocator->_thread_nodes, sizeof(uint32_t *),
        hipMemcpyDeviceToHost));
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_idxes, &d_allocator->_idxes, sizeof(uint32_t *), hipMemcpyDeviceToHost));
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_thread_tails, &d_allocator->_thread_tails, sizeof(uint32_t *),
        hipMemcpyDeviceToHost));

    CUDA_ASSERT_SUCCESS(hipFree(d_idxes));
    CUDA_ASSERT_SUCCESS(hipFree(d_thread_tails));
    CUDA_ASSERT_SUCCESS(hipFree(d_data));
    CUDA_ASSERT_SUCCESS(hipFree(d_node_counters));
    CUDA_ASSERT_SUCCESS(hipFree(d_thread_nodes));
    CUDA_ASSERT_SUCCESS(hipFree(d_allocator));
}

// ------------------------------
// GPU functions
// ------------------------------

void cuda_Allocator::ConsolidateHost(const uint32_t t_idx, std::barrier<> &barrier, bool isLastRun) {
    if (isLastRun) {
        /* reset node counters for others and leave */
        _node_counters[t_idx] = _max_node_per_thread + 1;

        /* mark this thread as inactive */
        _thread_nodes[t_idx] = 0;

        if (t_idx == _cleanup_thread) {
            /* wait for all threads to finish using allocator and possibly leave */
            barrier.arrive_and_wait();

            /* Perform last cleaning */
            _prepareIdxes();

            /* if this is a cleanup thread move ownership to the next thread */
            /* we are sure all threads that must be done are done at this moment and all other threads will rerun this function */
            for (size_t idx = 0; idx < _max_threads; ++idx) {
                if (_thread_nodes[idx] != 0) {
                    _cleanup_thread = idx;
                    break;
                }
            }
        }

        barrier.arrive_and_drop();
        return;
    }

    /* wait for all threads to finish using allocator */
    barrier.arrive_and_wait();

    /* first thread will update global data */
    if (t_idx == _cleanup_thread) {
        _prepareIdxes();
    }

    /* wait for first thread to update global data */
    barrier.arrive_and_wait();

    /* each of threads will clean up its allocator space */
    _cleanUpOwnSpace(t_idx);
}

// ------------------------------
// Cuda data functions
// ------------------------------

cuda_Data::cuda_Data(const BinSequencePack &pack): cuda_Data(pack.sequences.size(),
                                                             (pack.max_seq_size_bits + 63) / 32) {
    std::cout << "Max sequence size: " << pack.max_seq_size_bits << std::endl;

    static constexpr uint64_t kBitMask32 = ~static_cast<uint32_t>(0);

    for (size_t seq_idx = 0; seq_idx < pack.sequences.size(); ++seq_idx) {
        const auto &sequence = pack.sequences[seq_idx];
        auto fetcher = (*this)[seq_idx];
        fetcher.GetSequenceLength() = sequence.GetSizeBits();

        /* user dwords for better performance */
        for (size_t qword_idx = 0; qword_idx < sequence.GetSizeWords(); ++qword_idx) {
            const uint64_t qword = sequence.GetWord(qword_idx);
            const size_t dword_idx = qword_idx * 2;
            const uint32_t lo = qword & kBitMask32;
            const uint32_t hi = (qword >> 32) & kBitMask32;

            fetcher.GetWord(dword_idx) = lo;
            fetcher.GetWord(dword_idx + 1) = hi;
        }
    }
}

cuda_Data *cuda_Data::DumpToGPU() const {
    /* allocate manager object */
    cuda_Data *d_data;

    CUDA_ASSERT_SUCCESS(hipMalloc(&d_data, sizeof(cuda_Data)));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_data, this, sizeof(cuda_Data), hipMemcpyHostToDevice));

    /* allocate data itself */
    uint32_t *d_data_data;

    const size_t data_size = _num_sequences_padded32 * (_max_sequence_length + 1) * sizeof(uint32_t);
    CUDA_ASSERT_SUCCESS(hipMalloc(&d_data_data, data_size));
    CUDA_ASSERT_SUCCESS(hipMemcpy(d_data_data, _data, data_size, hipMemcpyHostToDevice));

    /* update manager object */
    CUDA_ASSERT_SUCCESS(hipMemcpy(&d_data->_data, &d_data_data, sizeof(uint32_t *), hipMemcpyHostToDevice));

    return d_data;
}

uint32_t *cuda_Data::GetDataPtrHost(const cuda_Data *d_data) {
    uint32_t *ptr;
    CUDA_ASSERT_SUCCESS(hipMemcpy(&ptr, &d_data->_data, sizeof(uint32_t *), hipMemcpyDeviceToHost));
    return ptr;
}

void cuda_Data::DeallocGPU(cuda_Data *d_data) {
    uint32_t *d_data_ptr = GetDataPtrHost(d_data);
    CUDA_ASSERT_SUCCESS(hipFree(d_data_ptr));
    CUDA_ASSERT_SUCCESS(hipFree(d_data));
}
