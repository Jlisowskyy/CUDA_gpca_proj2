#include "hip/hip_runtime.h"
#include <global_conf.cuh>
#include <defines.cuh>
#include <allocators.cuh>

#include <iostream>

cuda_GlobalConf *g_cudaGlobalConf{};

void cuda_InitGlobalConf() {
    g_cudaGlobalConf = new cuda_GlobalConf{};

    CUDA_ASSERT_SUCCESS(hipStreamCreate(&g_cudaGlobalConf->asyncStream));

    // /* query free memory */

    // size_t free, total;
    // CUDA_ASSERT_SUCCESS(hipMemGetInfo(&free, &total));
    //
    // /* set heap size to 80% of free memory */
    // const size_t heap_size = std::max(static_cast<size_t>(0), (8 * free / 10) - kPageSize);
    // CUDA_ASSERT_SUCCESS(hipDeviceSetLimit(hipLimitMallocHeapSize, heap_size));
    //
    // std::cout << "Set CUDA heap size to " << heap_size << " bytes, MB: " << heap_size / 1024 / 1024 << std::endl;
}

void cuda_DestroyGlobalConf() {
    CUDA_ASSERT_SUCCESS(hipStreamDestroy(g_cudaGlobalConf->asyncStream));

    delete g_cudaGlobalConf;
    g_cudaGlobalConf = nullptr;
}
