#include "hip/hip_runtime.h"
/* internal includes */
#include <thread_mgr.cuh>
#include <defines.cuh>

/* external includes */
#include <vector>
#include <iostream>

// ------------------------------
// Helpers
// ------------------------------

static constexpr uint32_t pow2(const uint32_t pow) {
    if (pow == 0) {
        return 1;
    }

    return 2 * pow2(pow - 1);
}

static constexpr uint32_t GenMask(const uint32_t size) {
    uint32_t mask{};

    for (uint32_t i = 0; i < size; ++i) {
        mask |= static_cast<uint32_t>(1) << i;
    }

    return mask;
}

// ------------------------------
// Constants
// ------------------------------

/* TODO: replace with bucket search logic */
static constexpr uint32_t kPrefixSize = 15;

// ------------------------------
// Implementations
// ------------------------------

MgrTrieBuildData ThreadMgr::PrepareTrieBuildData(const BinSequencePack &pack) const {
    /* TODO: adjust */
    static constexpr uint32_t kMaxThreads = pow2(kPrefixSize);

    /* TODO: adjust */
    static constexpr uint32_t kNumThreadsPerBlock = 512;

    MgrTrieBuildData data{};

    /* fill allocator management */
    data.max_nodes = pack.sequences.size() * pack.max_seq_size_bits;
    data.max_threads = kMaxThreads;
    data.max_nodes_per_thread = pack.max_seq_size_bits;

    /* fill trie kernel management */
    data.num_threads_per_block = kNumThreadsPerBlock;
    data.num_blocks = data.max_threads / data.num_threads_per_block;

    _prepareBuckets(pack, data);

    return data;
}

MgrTrieSearchData ThreadMgr::PrepareSearchData() const {
    /* TODO: replace with some logic */
    static constexpr uint32_t kMaxSolutions = 1'000'00;

    /* TODO: adjust */
    static constexpr uint32_t kThreadsPerBlock = 1024;

    /* TODO: adjust */
    static constexpr uint32_t kMaxBlocks = 1024;

    MgrTrieSearchData data{};

    data.num_solutions = kMaxSolutions;
    data.num_threads_per_block = kThreadsPerBlock;
    data.num_blocks = kMaxBlocks;

    return data;
}

void ThreadMgr::_prepareBuckets(const BinSequencePack &pack, MgrTrieBuildData &data) const {
    static constexpr double kMaxDeviation = 0.3;

    /* TODO: replace */
    static constexpr uint32_t kPrefixMask = GenMask(kPrefixSize);

    std::vector<uint32_t> prefixes{};
    prefixes.resize(data.max_threads);

    for (size_t t_idx = 0; t_idx < data.max_threads; ++t_idx) {
        /* TODO: */
        prefixes[t_idx] = kPrefixSize;
    }

    std::vector<std::vector<uint32_t> > buckets{};
    buckets.resize(data.max_threads);

    for (uint32_t seq_idx = 0; seq_idx < pack.sequences.size(); ++seq_idx) {
        /* TODO: */
        const auto &seq = pack.sequences[seq_idx];
        const auto idx = seq.GetWord(0) & kPrefixMask;

        buckets[idx].push_back(seq_idx);
    }

    /* TODO: */
    data.build_on_device = false;
    return;

    /* verify standard deviation */
    const auto [std_dev, max_occup] = _inspectBuckets(buckets);

    std::cout << "Acquired standard deviation: " << std_dev << std::endl;

    if (std_dev > kMaxDeviation) {
        std::cout << "Standard deviation is too high!" << std::endl;
        std::cout << "Fallback to cpu algorithm" << std::endl;
        data.build_on_device = false;
        return;
    }

    std::cout << "Standard deviation is acceptable proceeding to GPU TRIE build..." << std::endl;
    data.build_on_device = true;

    _dumpBucketsToGpu(buckets, prefixes, data, max_occup);
}

std::tuple<double, uint32_t> ThreadMgr::_inspectBuckets(const std::vector<std::vector<uint32_t> > &buckets) const {
    uint32_t sum{};
    uint32_t max_bucket{};
    for (const auto &bucket: buckets) {
        sum += bucket.size();
        max_bucket = std::max(max_bucket, static_cast<uint32_t>(bucket.size()));
    }

    const double mean = static_cast<double>(sum) / buckets.size();
    double dev_sum{};
    for (const auto &bucket: buckets) {
        const double a = static_cast<double>(bucket.size()) - mean;
        dev_sum += a * a;
    }
    const double std_dev = sqrt(dev_sum / buckets.size());

    return {std_dev, max_bucket};
}

void ThreadMgr::_dumpBucketsToGpu(const std::vector<std::vector<uint32_t> > &buckets,
                                  const std::vector<uint32_t> &prefixes, MgrTrieBuildData &data,
                                  uint32_t max_occup) const {
    /* Format data for GPU */
    // uint32_t &counter = buckets[idx];
    // buckets[(1 + counter) * data.max_threads + idx] = seq_idx;
    // ++counter;

    /* transfer to device */
    // CUDA_ASSERT_SUCCESS(hipMalloc(&data.d_buckets, buckets.size() * sizeof(uint32_t)));
    // CUDA_ASSERT_SUCCESS(hipMemcpy(data.d_buckets, buckets.data(), buckets.size() * sizeof(uint32_t),
    //     hipMemcpyHostToDevice));
    //
    // CUDA_ASSERT_SUCCESS(hipMalloc(&data.d_bucket_prefix_len, prefixes.size() * sizeof(uint32_t)));
    // CUDA_ASSERT_SUCCESS(hipMemcpy(data.d_bucket_prefix_len, prefixes.data(), prefixes.size() * sizeof(uint32_t),
    //     hipMemcpyHostToDevice));
}
